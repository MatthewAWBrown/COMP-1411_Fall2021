#include "hip/hip_runtime.h"
﻿/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 */

/************************************************************************
* COMP1411 - Computer Programming I                                     *
* Lab 10                                                                *
* Matthew Brown, Student ID# 1126992                                    *
************************************************************************/

#include <stdio.h>
#include <time.h>


  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector multiplication of A and B into C.
 * The 3 vectors have the same number of elements numElements.
 */
__global__
void vectorMUL(const float* A, const float* B, float* C, unsigned long numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] * B[i];
    }
}

void checkErr(hipError_t err, const char* msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 100000000;
    if (argc == 2) {
        numElements = strtoul(argv[1], 0, 10);
    }
    size_t size = numElements * sizeof(float);
    printf("[Vector multiplication of %lu elements]\n", numElements);

    // Allocate the host input vectors A & B
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);

    // Allocate the host output vector C
    float* h_C = (float*)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    //start cuda timer
    clock_t cudastart_t = clock();

    // 1a. Allocate the device input vectors A & B
    float* d_A = NULL;
    err = hipMalloc((void**)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");
    float* d_B = NULL;
    err = hipMalloc((void**)&d_B, size);
    checkErr(err, "Failed to allocate device vector B");

    // 1.b. Allocate the device output vector C
    float* d_C = NULL;
    err = hipMalloc((void**)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
    clock_t hosttstart_t = clock();

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");


    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector B from host to device");

    clock_t hosttend_t = clock();
    double hosttotal_t = (double)(hosttend_t - hosttstart_t) / CLOCKS_PER_SEC;
    printf("Host-to-device transfer timer: %LF\n", hosttotal_t);

    // 3. Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorMUL << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorAdd kernel");

    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    clock_t devtstart_t = clock();
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy vector C from device to host");
    clock_t devtend_t = clock();
    double devtotal_t = (double)(devtend_t - devtstart_t) / CLOCKS_PER_SEC;
    printf("Device-to-host transfer time: %lf\n", devtotal_t);

    //stop cuda timer
    clock_t cudaend_t = clock();
    //get total cuda time
    double cudatotal_t = (double)(cudaend_t - cudastart_t) / CLOCKS_PER_SEC;

    //start cpu timer
    clock_t start_t = clock();
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] * h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }



    printf("CUDA test PASSED\n");
    printf("CUDA time: %LF\n", cudatotal_t); 

        // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_B);
    checkErr(err, "Failed to free device vector B");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");


    // repeat the computation sequentially
    for (int i = 0; i < numElements; ++i)
    {
        h_C[i] = h_A[i] * h_B[i];
    }

    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] * h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    //stop timer
    clock_t end_t = clock();
    //get total time
    double total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;

    printf("\nNormal test PASSED\n");
    printf("Normal time: %LF\n", total_t); 

        // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");
    return 0;
}
